
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>
#include "assert.h"

// implicit constraint:
// tblock_tile_m * tblock_tile_k = 4 * warp_size * tblock_size_m * tblock_size_n


#define smem_switch_a 0x2000
#define smem_switch_b 0x1000

// seems that this have to be 2^k??
// 20000 x 32 x 32
/*
#define smem_switch_a (16 * 1024)
#define smem_switch_b 1024
*/

#define warp_size 32
// default values: 128, 128, 8. to try 64, 128, 4


// 20000 x 32 x 32
/*
#define tblock_tile_m 256
#define tblock_tile_n 32
#define tblock_tile_k 8
*/
// 5000 x 32 x 32
/*
#define tblock_tile_m 128
#define tblock_tile_n 32
#define tblock_tile_k 4
*/
// 5000 x 128 x 128
/*
#define tblock_tile_m 128
#define tblock_tile_n 128
#define tblock_tile_k 4
*/
#define tblock_tile_m 128
#define tblock_tile_n 128
#define tblock_tile_k 8

/*
#define warp_tile_m 32
#define warp_tile_n 64
*/
#define thread_tile_m 8
#define thread_tile_n 8
// 20000 x 32 x 32
/*
#define thread_ldg_a 16
#define thread_ldg_b 2
*/
// 5000 x 32 x 32
/*
#define thread_ldg_a 8
#define thread_ldg_b 2
*/
// 5000 x 128 x 128
/*
#define thread_ldg_a 2
#define thread_ldg_b 2
*/
#define thread_ldg_a 4
#define thread_ldg_b 4

// 5000/20000 x 32 x 32
/*
#define warp_tile_m 64
#define warp_tile_n 32
*/
// 5000 x 128 x 128
#define warp_tile_m 32
#define warp_tile_n 64

#define warp_size_m (warp_tile_m/thread_tile_m)
#define warp_size_n (warp_tile_n/thread_tile_n)
#define tblock_size_m (tblock_tile_m/warp_tile_m)
#define tblock_size_n (tblock_tile_n/warp_tile_n)

// warp_tile_m / thread_tile_m
/*
#define warp_size_m 4
// warp_tile_n / thread_tile_n
#define warp_size_n 8
// tblock_tile_m / warp_tile_m
#define tblock_size_m 4
// tblock_tile_n / warp_tile_n
#define tblock_size_n 2
*/

/*
#define warp_tile_m 64
#define warp_tile_n 32
#define warp_size_m 8
#define warp_size_n 4
#define tblock_size_m 2
#define tblock_size_n 4
*/

/*
#define warp_tile_m 128
#define warp_tile_n 16
#define warp_size_m 16
#define warp_size_n 2
#define tblock_size_m 1
#define tblock_size_n 8
*/



void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}

bool check(const float *A,
           const float *B,
           const float *C,
           int m, int n, int k) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.f;
            for (int p = 0; p < k; ++p) {
                sum += A[i * k + p] * B[j + p * n];
            }

            if (std::fabs(sum - C[i * n + j]) / std::fabs(sum) > 1e-5f) {
                printf("C[%d][%d] not match, %f vs %f\n", i, j, sum, C[i * n + j]);
                return false;
            }
        }
    }

    return true;
}

__device__ __forceinline__
uint32_t smem_u32addr(const void *smem_ptr) {
    uint32_t addr;
    asm ("{.reg .u64 u64addr;\n"
         " cvta.to.shared.u64 u64addr, %1;\n"
         " cvt.u32.u64 %0, u64addr;}\n"
         : "=r"(addr)
         : "l"(smem_ptr)
    );

    return addr;
}

__device__ __forceinline__
void ldg32_nc(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void ldg32_nc_0(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @!p mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void stg32(const float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @p st.global.f32 [%0], %1;}\n"
        : : "l"(ptr), "f"(reg), "r"((int)guard)
    );
}

__device__ __forceinline__
void lds128(float &reg0, float &reg1,
            float &reg2, float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
        : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
        : "r"(addr)
    );
}

__device__ __forceinline__
void sts32(const float &reg, const uint32_t &addr) {
    asm volatile (
        "st.shared.f32 [%0], %1;\n"
        : : "r"(addr), "f"(reg)
    );
}

__device__ __forceinline__
void sts128(const float &reg0, const float &reg1,
            const float &reg2, const float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3)
    );
}

__device__ __forceinline__
void sts64(const float &reg0, const float &reg1,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v2.f32 [%0], {%1, %2};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1)
    );
}

struct StgFrag {
    float data[4][4];

    __device__ __forceinline__
    StgFrag(const float (&C_frag)[8][8], int tile_x, int tile_y) {
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                data[i][j] = C_frag[tile_y * 4 + i][tile_x * 4 + j];
            }
        }
    }
};

__device__ __noinline__
void C_tile_wb(StgFrag C_frag,
               float *C_stg_ptr,
               const float *C_lds_ptr,
               uint32_t C_sts_addr,
               uint32_t m,
               uint32_t n,
               uint32_t m_idx,
               uint32_t n_idx) {
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        sts128(C_frag.data[i][0],
               C_frag.data[i][1],
               C_frag.data[i][2],
               C_frag.data[i][3],
               C_sts_addr + i * 8 * sizeof(float4));
    }

    __syncthreads();

    uint32_t m_guard = m < m_idx ? 0 : m - m_idx;

    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        stg32(C_lds_ptr[i * 32],
              C_stg_ptr + i * n,
              i < m_guard && n_idx < n);
    }
}

/*
 * matrix A, B and C: row-major
 *
 * mma block:
 * thread block tile: m128n128k8
 * warp tile: m32n64k8
 * thread tile: m8n8k8
 * thread fragment:
 *     matrixA: 8x1 FP32
 *     matrixB: 1x8 FP32
 *
 * ----------------------------------------------------------------
 * thread block tile map:
 *
 *                                128
 *                    --|---------------------|
 *             B_tile  8|                     |
 *                    --|---------------------|
 *
 *  A_tile   | 8 |      |    64    |
 *         --|---|    --|----------|----------|
 *           |   |    32|  warp_0  |  warp_1  |
 *           |   |    --|----------|----------|
 *           |   |      |  warp_2  |  warp_3  |
 *        128|   |      |----------|----------|
 *           |   |      |  warp_4  |  warp_5  |
 *           |   |      |----------|----------|
 *           |   |      |  warp_6  |  warp_7  |
 *         --|---|      |----------|----------|
 *
 * ----------------------------------------------------------------
 * warp tile map:
 *
 * 'z' thread map to avoid LDS.128 shared memory broadcast limitation.
 *
 *              |              32               ||
 *     B_frag --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 *             1|///|   |   |   |   |   |   |   ||///|   |   |   |   |   |   |   |
 *            --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 * A_frag       | 4 |                           ||
 *    | 1 |                                     ||
 *  --|---|--   |---|---|---|---|---|---|---|---||---|---------------------------|
 *    |///|4    |t0 |t2 |t4 |t6 |t8 |t10|t12|t14||t0 |                           |
 *    |---|--   |---|---|---|---|---|---|---|---||---|                           |
 *    |   |     |t1 |t3 |t5 |t7 |t9 |t11|t13|t15||                               |
 *  16|---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t16|t18|t20|t22|t24|t26|t28|t30||                               |
 *    |---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t17|t19|t21|t23|t25|t27|t29|t31||                               |
 *  ==|===|=====|===|===|===|===|===|===|===|===||===|============================
 *    |///|     |t0 |                           ||t0 |                           |
 *    |---|     |---|                           ||---|                           |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |-------------------------------||-------------------------------|
 *
 */

// haotian: seems that thread_tile can only be 8 = 4 (per thread load 4)x 2 (figure above)??

__global__ __launch_bounds__(256, 2)
void sgemm_128x128x8_kernel(const float *A,
                            const float *B,
                            float *C,
                            uint32_t m,
                            uint32_t n,
                            uint32_t k,
                            uint32_t A_ldg_step,    // k * sizeof(float)
                            uint32_t B_ldg_step) {  // n * sizeof(float) * tblock_tile_k
    /*
     * matrix A & B thread block tile shared memory (double buffer)
     * matrix A: 132 * 8 * 4Byte/item * double buffer = 4.125KB * 2
     * matrix B: 128 * 8 * 4Byte/item * double buffer = 8KB
     *
     * for double buffer faster switch, A_smem requires 8KB * 2 shared memory
     * and 16KB aligned, B_smem should be 8KB aligned, then the double buffer
     * can be switched by only 1 xor instruction:
     *     (uint32_t &)A_smem ^= 0x2000;
     *     (uint32_t &)B_smem ^= 0x1000;
     */
    
    // [!!] tbd: haotian: check whether the smem space is enough (20220102)
    /*
    __shared__ __align__(16 * 1024) char smem[24 * 1024];
    float *A_smem = reinterpret_cast<float *>(smem);
    float *B_smem = reinterpret_cast<float *>(smem + 16 * 1024);
    */
    __shared__ char smem[24 * 1024];
    float *A_smem = reinterpret_cast<float *>(smem);
    float *B_smem = reinterpret_cast<float *>(smem + 16 * 1024);
    
    // 20000 x 32 x 32, 5000 x 32 x 32, 5000 x 128 x 128
    /*
    __shared__ char smem[32 * 1024];
    float *A_smem = reinterpret_cast<float *>(smem);
    float *B_smem = reinterpret_cast<float *>(smem + 28 * 1024);
    */

    // A, B and C register fragment
    float A_frag[2][thread_tile_m];
    float B_frag[2][thread_tile_n];
    float C_frag[thread_tile_m][thread_tile_n];
    #pragma unroll
    for (int i = 0; i < thread_tile_m; ++i) {
        #pragma unroll
        for (int j = 0; j < thread_tile_n; ++j) {
            C_frag[i][j] = 0;
        }
    }

    const uint32_t lane_id = threadIdx.x % warp_size;
    const uint32_t warp_id = threadIdx.x / warp_size;

    // 4x8 threads each warp for FFMA
    
    /*
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);
    */
    // refer to the warp tile map figure!
    const uint32_t mma_tid_x = (lane_id / 2) % warp_size_n;
    // 2x16: lane_id/32, 4x8: /16, 8x4: /8, 16x2: /4, 32x1: /2
    const uint32_t mma_tid_y = (lane_id / (warp_size_n * 2)) * 2 + (lane_id % 2);
    
    //if(warp_id == 0 && blockIdx.x == 0 && blockIdx.y == 0) printf("%d %d %d\n", lane_id, mma_tid_x, mma_tid_y);

    // A_tile & B_tile ldg pointer
    // haotian: [1] equivalent to <<< (m/128, n/128), (32, 8)>>>, **each warp load 4** from DRAM to shared (tblock: 128x8).
    // haotian: [2] equivalent to <<< (m/128, n/128), (8, 32)>>>, **each warp load 4** from DRAM to shared.
    /*
    const char *A_ldg_ptr = (const char *)(
        A + (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8);
    
    const char *B_ldg_ptr = (const char *)(
        B + (threadIdx.x / 32) * n + blockIdx.x * 128 + threadIdx.x % 32);
    */
    // haotian: always make sure data loaded from gmem can be directly used (no need to wait for other warps).
    // eg. for A, warp 0 loaded 16x8 at 1st step. Note that later compute will first be using this 16 numbers. (16 x 2) x (32 x 2)
    const char *A_ldg_ptr = (const char *)(
        A + (blockIdx.y * tblock_tile_m + threadIdx.x / tblock_tile_k * thread_ldg_a) * k + threadIdx.x % tblock_tile_k);
    
    const char *B_ldg_ptr = (const char *)(
        B + (threadIdx.x / (tblock_tile_n / thread_ldg_b)) * n + blockIdx.x * tblock_tile_n + threadIdx.x % (tblock_tile_n / thread_ldg_b));

    // A_tile & B_tile sts/lds pointer
    // using uint32_t pointer for faster double buffer switch
    // haotian: [1] looks like 8 x 132 shared memory (what is 8x4 for?)
    // haotian: [2] looks like 128 x 8 shared memory
    // haotian: correlate with A_ldg_ptr and B_ldg_ptr, just transposed for A.
    /*
    uint32_t A_sts_addr = smem_u32addr(
        A_smem + (threadIdx.x % 8) * 132 + (threadIdx.x / 8) * 4);
    
    uint32_t B_sts_addr = smem_u32addr(
        B_smem + (threadIdx.x / 32) * 128 + (threadIdx.x % 32));
    */
    // haotian: +4 for efficiency. to be analyzed: why?
    uint32_t A_sts_addr = smem_u32addr( 
        A_smem + (threadIdx.x % tblock_tile_k) * (tblock_tile_m + 4) + (threadIdx.x / tblock_tile_k) * thread_ldg_a);
    uint32_t B_sts_addr = smem_u32addr(
        B_smem + (threadIdx.x / (tblock_tile_n / thread_ldg_b)) * tblock_tile_n + (threadIdx.x % (tblock_tile_n / thread_ldg_b)));
    
    // haotian: mma_tid_x in [0, 7] (for B shared mem), mma_tid_y in [0, 3] (for A shared mem), each thread load 4 (strided) x 4 (contig), there might be broadcast.
    // haotian: warp_id is 4x2. per warp 32x1 or 1x64
    /*
    uint32_t A_lds_addr = smem_u32addr(
        A_smem + (warp_id / 2) * 32 + mma_tid_y * 4);
    uint32_t B_lds_addr = smem_u32addr(
        B_smem + (warp_id % 2) * 64 + mma_tid_x * 4);
    */
    uint32_t A_lds_addr = smem_u32addr(
        A_smem + (warp_id / tblock_size_n) * warp_tile_m + mma_tid_y * 4);
    uint32_t B_lds_addr = smem_u32addr(
        B_smem + (warp_id % tblock_size_n) * warp_tile_n + mma_tid_x * 4);

    // ldg_guard to avoid LDG out of bound
    uint32_t A_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < thread_ldg_a; ++i) {
        /*
        int m_idx = blockIdx.y * 128 + threadIdx.x / 8 * 4 + i;
        */
        int m_idx = blockIdx.y * tblock_tile_m + threadIdx.x / tblock_tile_k * thread_ldg_a + i;
        if (m_idx < m) {
            A_ldg_guard |= (1u << i);
        }
    }

    uint32_t B_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < thread_ldg_b; ++i) {
        /*
        int n_idx = blockIdx.x * 128 + threadIdx.x % 32 + i * 32;
        */
        int n_idx = blockIdx.x * tblock_tile_n + threadIdx.x % (tblock_tile_n / thread_ldg_b) + i * (tblock_tile_n / thread_ldg_b);
        if (n_idx < n) {
            B_ldg_guard |= (1u << i);
        }
    }

    float A_ldg_reg[thread_ldg_a];
    float B_ldg_reg[thread_ldg_b];

    // 1'st A&B tile loaded before the k_tile loop
    /*
    uint32_t k_tiles = (k + 7) / 8 - 1;
    */
    uint32_t k_tiles = (k + tblock_tile_k - 1) / tblock_tile_k - 1;

    // load 1'st tile to shared memory
    // haotian: A load 128x1, B load 1x128, each thread load 4.
    // a stored as 1x128, B stored as 128x1
    {
        /*
        uint32_t first_k_tile = k - k_tiles * 8;
        */
        uint32_t first_k_tile = k - k_tiles * tblock_tile_k;

        #pragma unroll
        for (int i = 0; i < thread_ldg_a; ++i) {
            /*
            bool guard = (A_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x % 8 < first_k_tile;
            */
            bool guard = (A_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x % tblock_tile_k < first_k_tile;
            
            ldg32_nc_0(A_ldg_reg[i],
                       A_ldg_ptr + i * A_ldg_step,
                       guard);
        }

        if constexpr (thread_ldg_a > 4){
            #pragma unroll
            for(int i = 0; i < thread_ldg_a / 4; i++){
                sts128(A_ldg_reg[i * 4], A_ldg_reg[i * 4 + 1], A_ldg_reg[i * 4 + 2], A_ldg_reg[i * 4 + 3],
                    A_sts_addr + i * 16);
            }
        }
        else if (thread_ldg_a == 4){
            sts128(A_ldg_reg[0], A_ldg_reg[1], A_ldg_reg[2], A_ldg_reg[3],
                A_sts_addr);
        }
        else if (thread_ldg_a == 2){
            sts64(A_ldg_reg[0], A_ldg_reg[1], A_sts_addr);
        }
        else{
            sts32(A_ldg_reg[0], A_sts_addr);
        }

        #pragma unroll
        for (int i = 0; i < thread_ldg_b; ++i) {
            /*
            bool guard = (B_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x / 32 < first_k_tile;
            ldg32_nc_0(B_ldg_reg[i],    
                       B_ldg_ptr + i * 32 * sizeof(float),
                       guard);
            */
            bool guard = (B_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x / (tblock_tile_n / thread_ldg_b) < first_k_tile;
            ldg32_nc_0(B_ldg_reg[i],    
                       B_ldg_ptr + i * (tblock_tile_n / thread_ldg_b) * sizeof(float),
                       guard);
        }

        #pragma unroll
        for (int i = 0; i < thread_ldg_b; ++i) {
            /*
            sts32(B_ldg_reg[i], B_sts_addr + i * 32 * sizeof(float));
            */
            sts32(B_ldg_reg[i], B_sts_addr + i * (tblock_tile_n / thread_ldg_b) * sizeof(float));
        }

        __syncthreads();

        // switch double buffer. TBD: these two numbers are correlated with 128x8 tile size
        A_sts_addr ^= smem_switch_a;
        B_sts_addr ^= smem_switch_b;

        // ldg pointer for next tile
        A_ldg_ptr += first_k_tile * sizeof(float);
        B_ldg_ptr += n * first_k_tile * sizeof(float);
    }

    // load 1'st fragment
    // haotian: 16 and 32: see the figure above. Assume each thread loads 4.
    /*
    lds128(A_frag[0][0], A_frag[0][1], A_frag[0][2], A_frag[0][3],
           A_lds_addr);
    lds128(A_frag[0][4], A_frag[0][5], A_frag[0][6], A_frag[0][7],
           A_lds_addr + 16 * sizeof(float));
    lds128(B_frag[0][0], B_frag[0][1], B_frag[0][2], B_frag[0][3],
           B_lds_addr);
    lds128(B_frag[0][4], B_frag[0][5], B_frag[0][6], B_frag[0][7],
           B_lds_addr + 32 * sizeof(float));
    */
    lds128(A_frag[0][0], A_frag[0][1], A_frag[0][2], A_frag[0][3],
           A_lds_addr);
    lds128(A_frag[0][4], A_frag[0][5], A_frag[0][6], A_frag[0][7],
           A_lds_addr + 4 * warp_size_m * sizeof(float));
    lds128(B_frag[0][0], B_frag[0][1], B_frag[0][2], B_frag[0][3],
           B_lds_addr);
    lds128(B_frag[0][4], B_frag[0][5], B_frag[0][6], B_frag[0][7],
           B_lds_addr + 4 * warp_size_n * sizeof(float));


    // k_tiles loop
    for (; k_tiles > 0; --k_tiles) {
        #pragma unroll
        for(int k_frag = 0; k_frag < tblock_tile_k; ++k_frag){
        //for (int k_frag = 0; k_frag < 8; ++k_frag) {
            // store next A&B tile to shared memory
            // Haotian: note: there is one tile not used! so we still need an epilog.
            if (k_frag == tblock_tile_k - 1){
            //if (k_frag == 7) {
                if constexpr (thread_ldg_a > 4){
                    #pragma unroll
                    for(int i = 0; i < thread_ldg_a / 4; i++){
                        sts128(A_ldg_reg[i * 4], A_ldg_reg[i * 4 + 1], A_ldg_reg[i * 4 + 2], A_ldg_reg[i * 4 + 3],
                            A_sts_addr + i * 16);
                    }
                }
                else if (thread_ldg_a == 4){
                    sts128(A_ldg_reg[0], A_ldg_reg[1], A_ldg_reg[2], A_ldg_reg[3],
                        A_sts_addr);
                }
                else if (thread_ldg_a == 2){
                    sts64(A_ldg_reg[0], A_ldg_reg[1], A_sts_addr);
                }
                else{
                    sts32(A_ldg_reg[0], A_sts_addr);
                }
                #pragma unroll
                for (int i = 0; i < thread_ldg_b; ++i) {
                    /*
                    sts32(B_ldg_reg[i], B_sts_addr + i * 32 * sizeof(float));
                    */
                    sts32(B_ldg_reg[i], B_sts_addr + i * (tblock_tile_n / thread_ldg_b) * sizeof(float));
                }

                __syncthreads();

                // switch double buffer
                A_lds_addr ^= smem_switch_a;
                B_lds_addr ^= smem_switch_b;
                A_sts_addr ^= smem_switch_a;
                B_sts_addr ^= smem_switch_b;

                // ldg pointer for next tile
                /*
                A_ldg_ptr += 8 * sizeof(float);
                */
                A_ldg_ptr += tblock_tile_k * sizeof(float);
                // haotian: B_ldg_step = 8 * n * sizeof(float). n dimension tiled over tblock.
                B_ldg_ptr += B_ldg_step;
            }

            // load next A&B fragment from shared memory to register. Haotian: naturally double buffer (k_flag switches between even and odd)
            // +16: each thread load 4 x warp_size_m 4 = 16, +32: each thread load 4 x warp_size_n 8 = 32
            /*
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float));
            */
            
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % tblock_tile_k * (tblock_tile_m + 4) * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % tblock_tile_k * (tblock_tile_m + 4) + 4 * warp_size_m) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % tblock_tile_k * tblock_tile_n * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % tblock_tile_k * tblock_tile_n + 4 * warp_size_n) * sizeof(float));
            
            // load next A&B tile. Haotian: 4 = 128 (tblock tile size) / 32 ??
            if (k_frag == 0) {
                #pragma unroll
                for (int i = 0; i < thread_ldg_a; ++i) {
                    ldg32_nc(A_ldg_reg[i],
                             A_ldg_ptr + i * A_ldg_step,
                             (A_ldg_guard & (1u << i)) != 0);
                }

                #pragma unroll
                for (int i = 0; i < thread_ldg_b; ++i) {
                    /*
                    ldg32_nc(B_ldg_reg[i],
                             B_ldg_ptr + i * 32 * sizeof(float),
                             (B_ldg_guard & (1u << i)) != 0);
                    */
                    ldg32_nc(B_ldg_reg[i],
                             B_ldg_ptr + i * (tblock_tile_n / thread_ldg_b) * sizeof(float),
                             (B_ldg_guard & (1u << i)) != 0);
                }
            }

            // FFMA loop
            /*
            #pragma unroll
            for (int i = 0; i < 8; ++i) {
                #pragma unroll
                for (int j = 0; j < 8; ++j) {
                    C_frag[i][j] += A_frag[k_frag % 2][i] *
                                    B_frag[k_frag % 2][j];
                }
            }
            */
            #pragma unroll
            for (int i = 0; i < thread_tile_m; ++i) {
                #pragma unroll
                for (int j = 0; j < thread_tile_n; ++j) {
                    C_frag[i][j] += A_frag[k_frag % 2][i] *
                                    B_frag[k_frag % 2][j];
                }
            }

        }
    }

    // FFMA for the last tile
    #pragma unroll
    //for (int k_frag = 0; k_frag < 8; ++k_frag) {
    //    if (k_frag < 7) {
    for (int k_frag = 0; k_frag < tblock_tile_k; ++k_frag) {
        if (k_frag < tblock_tile_k - 1) {
            // load next A&B fragment from shared memory to register
            /*
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % 8 * 132 * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % 8 * 132 + 16) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float));
            */
            
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % tblock_tile_k * (tblock_tile_m + 4) * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % tblock_tile_k * (tblock_tile_m + 4) + 4 * warp_size_m) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % tblock_tile_k * tblock_tile_n * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % tblock_tile_k * tblock_tile_n + 4 * warp_size_n) * sizeof(float));
        }

        // FFMA loop
        /*
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            #pragma unroll
            for (int j = 0; j < 8; ++j) {
                C_frag[i][j] += A_frag[k_frag % 2][i] *
                                B_frag[k_frag % 2][j];
            }
        }
        */
        #pragma unroll
        for (int i = 0; i < thread_tile_m; ++i) {
            #pragma unroll
            for (int j = 0; j < thread_tile_n; ++j) {
                C_frag[i][j] += A_frag[k_frag % 2][i] *
                                B_frag[k_frag % 2][j];
            }
        }
    }

    // C_tile write back, reuse A&B tile shared memory buffer
    // haotian: mma_tid_x <= 8, mma_tid_y <= 4, per thread loads 4 data
    
    // [!!] tbd; haotian: in the mma_tid_x direction, since float4, we dont need to x4, but x4 still needed on mma_tid_y direction.
    
    /*
    uint32_t C_sts_addr = smem_u32addr((float4 *)(smem + warp_id * 2048) +
                                       mma_tid_y * 4 * 8 + mma_tid_x);
    const float *C_lds_ptr = (float *)(smem + warp_id * 2048) + lane_id;
    */
    uint32_t C_sts_addr = smem_u32addr((float4 *)(smem + warp_id * (warp_tile_m * warp_tile_n)) +
                                       mma_tid_y * 4 * warp_size_n + mma_tid_x);
    const float *C_lds_ptr = (float *)(smem + warp_id * (warp_tile_m * warp_tile_n)) + lane_id;

    /*
    uint32_t m_idx = blockIdx.y * 128 + warp_id / 2 * 32;
    uint32_t n_idx = blockIdx.x * 128 + warp_id % 2 * 64 + lane_id;
    */
    uint32_t m_idx = blockIdx.y * tblock_tile_m + warp_id / tblock_size_n * warp_tile_m + lane_id / (4 * warp_size_n);
    uint32_t n_idx = blockIdx.x * tblock_tile_n + warp_id % tblock_size_n * warp_tile_n + lane_id % (4 * warp_size_n);
    // haotian: this might be problematic when warp_tile_n != 64.
    float *C_stg_ptr = C + m_idx * n + n_idx;

    if (m_idx >= m) {
        return;
    } else if (m_idx + warp_tile_m <= m) {
        // haotian: 32 = warp_tile_m

        // haotian: each 4x4 is contiguous. i bound 2 = thread tile / 4, sim. for j bound.
        // haotian: store to smem contiguously. 
        // haotian: according to the figure above (reg -> smem transfer). Note: space reused for different i,j.
        // haotian: smem->gmem: each thread store 1 number, 32 threads access memory contiguously. store 16x32 once.
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;
        /*
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts128(C_frag[i * 4 + p][j * 4],
                           C_frag[i * 4 + p][j * 4 + 1],
                           C_frag[i * 4 + p][j * 4 + 2],
                           C_frag[i * 4 + p][j * 4 + 3],
                           C_sts_addr + p * 8 * sizeof(float4));
                }

                __syncthreads();
                #pragma unroll
                for (int p = 0; p < 16; ++p) {
                    stg32(C_lds_ptr[p * 32],
                          C_stg_ptr + (i * 16 + p) * n + j * 32,
                          j * 32 < n_guard);
                }
            }
        }
        */
        #pragma unroll
        for (int i = 0; i < thread_tile_m / 4; ++i) {
            #pragma unroll
            for (int j = 0; j < thread_tile_n / 4; ++j) {
                __syncthreads();
                // store a 4x4 for each thread, each warp: 16x32 in default case.
                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts128(C_frag[i * 4 + p][j * 4],
                           C_frag[i * 4 + p][j * 4 + 1],
                           C_frag[i * 4 + p][j * 4 + 2],
                           C_frag[i * 4 + p][j * 4 + 3],
                           C_sts_addr + p * warp_size_n * sizeof(float4));
                }

                __syncthreads();
                // haotian: each warp stores 512 = 32 x (4 x 4) elements in one inner-loop step
                // haotian: currently only solves warp_tile_n > 32, but what about warp_tile_n < 32? q?? (solved)
                // haotian: to solve slice_k, need to perform reduction here.
                if constexpr (warp_size_n == 8){
                    #pragma unroll
                    for (int p = 0; p < 16; ++p) {
                        stg32(C_lds_ptr[p * 32],
                            C_stg_ptr + (i * 16 + p) * n + j * 32,
                            j * 32 < n_guard);
                    }
                }
                else if (warp_size_n > 8){
                    #pragma unroll
                    for(int p = 0; p < 4 * warp_size_m; ++p){
                        #pragma unroll
                        for(int q = 0; q < warp_size / 8; ++q){
                            stg32(C_lds_ptr[p * 4 * warp_size_n + q * 32],
                                C_stg_ptr + (i * 4 * warp_size_m + p) * n + j * 4 * warp_size_n + q * 32,
                                (j * 4 * warp_size_n + q * 32 < n_guard));
                        }
                    }
                }
                else{
                    #pragma unroll
                    for (int p = 0; p < 16; ++p) {
                        //printf("%d %d %d %d\n", (i * 16 + p) * (8 / warp_size_n), j * 4 * warp_size_n, m_idx, n_idx);
                        stg32(C_lds_ptr[p * 32],
                            C_stg_ptr + (i * 16 + p) * (8 / warp_size_n) * n + j * 4 * warp_size_n,
                            (j * 4 * warp_size_n < n_guard));
                    }
                }
            }
        }
    } /*else {
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                StgFrag stg_frag(C_frag, j, i);

                C_tile_wb(stg_frag,
                          C_stg_ptr + i * 16 * n + j * 32,
                          C_lds_ptr,
                          C_sts_addr,
                          m,
                          n,
                          m_idx + i * 16,
                          n_idx + j * 32);
            }
        }
    }*/
    else{
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;
        uint32_t m_guard = m < m_idx ? 0 : m - m_idx;
        // haotian: just copy the previous code, branch predictor issue
        #pragma unroll
        for (int i = 0; i < thread_tile_m / 4; ++i) {
            #pragma unroll
            for (int j = 0; j < thread_tile_n / 4; ++j) {
                __syncthreads();
                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts128(C_frag[i * 4 + p][j * 4],
                           C_frag[i * 4 + p][j * 4 + 1],
                           C_frag[i * 4 + p][j * 4 + 2],
                           C_frag[i * 4 + p][j * 4 + 3],
                           C_sts_addr + p * warp_size_n * sizeof(float4));
                }

                __syncthreads();
                if constexpr (warp_size_n == 8){
                    #pragma unroll
                    for (int p = 0; p < 16; ++p) {
                        stg32(C_lds_ptr[p * 32],
                            C_stg_ptr + (i * 16 + p) * n + j * 32,
                            i * 16 + p < m_guard && j * 32 < n_guard);
                    }
                }
                else if (warp_size_n > 8){
                    #pragma unroll
                    for(int p = 0; p < 4 * warp_size_m; ++p){
                        #pragma unroll
                        for(int q = 0; q < warp_size / 8; ++q){
                            stg32(C_lds_ptr[p * 4 * warp_size_n + q * 32],
                                C_stg_ptr + (i * 4 * warp_size_m + p) * n + j * 4 * warp_size_n + q * 32,
                                (i * 4 * warp_size_m + p < m_guard) && (j * 4 * warp_size_n + q * 32 < n_guard));
                        }
                    }
                }
                else{
                    #pragma unroll
                    for (int p = 0; p < 16; ++p) {
                        //printf("%d %d %d %d\n", (i * 16 + p) * (8 / warp_size_n), j * 4 * warp_size_n, m_idx, n_idx);
                        stg32(C_lds_ptr[p * 32],
                            C_stg_ptr + (i * 16 + p) * (8 / warp_size_n) * n + j * 4 * warp_size_n,
                            ((i * 16 + p) * (8 / warp_size_n) < m_guard) && (j * 4 * warp_size_n < n_guard));
                    }
                }
            }
        }
    }

    //if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) printf("%f %f %f %f\n", C_frag[0][0], C_frag[1][0], C_frag[2][0], C_frag[3][0]);
    //if(threadIdx.x == 15 && blockIdx.x == 0 && blockIdx.y == 0) printf("%f %f %f %f %f\n", smem[0], smem[16], smem[32], smem[48], smem[64]);
}

int main() {
    /*
    int m = 5120;
    int n = 4096;
    int k = 4096;
    */
    /*
    int m = 20000;
    int n = 32;
    int k = 32;
    */
    int m = 5120;
    int n = 4096;
    int k = 4096;
    /*
    int m = 5000;
    int n = 128;
    int k = 128;
    */
    int n_iter = 10;
    // 1 warp = 32 threads constraint
    assert(warp_size_m * warp_size_n == warp_size);
    // need to load <= can load. problem why < cannot work: some warps do not load from gmem to smem,
    // but they need to use data from smem in ffma!
    printf(
        "A need to load: %d; B need to load: %d;\nA can load: %d; B can load: %d.\n", 
        tblock_tile_m * tblock_tile_k, tblock_tile_n * tblock_tile_k, thread_ldg_a * warp_size * tblock_size_m * tblock_size_n, thread_ldg_b * warp_size * tblock_size_m * tblock_size_n
    );
    assert(tblock_tile_m * tblock_tile_k == thread_ldg_a * warp_size * tblock_size_m * tblock_size_n);
    assert(tblock_tile_n * tblock_tile_k == thread_ldg_b * warp_size * tblock_size_m * tblock_size_n);

    float *h_A, *h_B, *h_C;

    hipHostMalloc(&h_A, m * k * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, k * n * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, m * n * sizeof(float), hipHostMallocDefault);
    random_init(h_A, m * k);
    random_init(h_B, k * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyDefault);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 grid((n + tblock_tile_n - 1) / tblock_tile_n, (m + tblock_tile_m - 1) / tblock_tile_m);
    
    //printf("%d %d %d\n", (n + tblock_tile_n - 1) / tblock_tile_n, (m + tblock_tile_m - 1) / tblock_tile_m, 32 * tblock_size_m * tblock_size_n);
    
    // warmup
    sgemm_128x128x8_kernel<<<grid, 32 * tblock_size_m * tblock_size_n>>>(
        d_A, d_B, d_C, m, n, k, k * sizeof(float), n * sizeof(float) * tblock_tile_k);

    hipEventRecord(start);
    for (int i = 0; i < n_iter; ++i) {
        sgemm_128x128x8_kernel<<<grid, 32 * tblock_size_m * tblock_size_n>>>(
            d_A, d_B, d_C, m, n, k, k * sizeof(float), n * sizeof(float) * tblock_tile_k);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    long workload = n_iter * long(m) * n * k * 2;
    double gflops = (double(workload) / 1e9) / (double(ms) / 1e3);
    printf("Performance: %fGFLOPS\n", gflops);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDefault);

    bool chk = check(h_A, h_B, h_C, m, n, k);
    printf("Matrix_C check: %s\n", chk ? "OK" : "Failed");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
}

