
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>

#define a_const 132

void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}

// Check the Results with naive implementation.
bool check(const float *A,
           const float *B,
           const float *C,
           int m, int n, int k) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.f;
            for (int p = 0; p < k; ++p) {
                sum += A[i * k + p] * B[j + p * n];
            }

            if (std::fabs(sum - C[i * n + j]) / std::fabs(sum) > 1e-5f) {
                printf("C[%d][%d] not match, %f vs %f\n", i, j, sum, C[i * n + j]);
                return false;
            }
        }
    }

    return true;
}

// ys: convert smem_ptr to unsigned 32 int for faster double buffer switch
__device__ __forceinline__
uint32_t smem_u32addr(const void *smem_ptr) {
    uint32_t addr;
    asm ("{.reg .u64 u64addr;\n"
         " cvta.to.shared.u64 u64addr, %1;\n"
         " cvt.u32.u64 %0, u64addr;}\n"
         : "=r"(addr)
         : "l"(smem_ptr)
    );

    return addr;
}

__device__ __forceinline__
void ldg32_nc(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void ldg32_nc_0(float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @!p mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 && \
    __CUDA_ARCH__ >= 750
        " @p ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
        " @p ld.global.nc.f32 %0, [%1];}\n"
#endif
        : "=f"(reg)
        : "l"(ptr), "r"((int)guard)
    );
}

__device__ __forceinline__
void stg32(const float &reg, const void *ptr, bool guard) {
    asm volatile (
        "{.reg .pred p;\n"
        " setp.ne.b32 p, %2, 0;\n"
        " @p st.global.f32 [%0], %1;}\n"
        : : "l"(ptr), "f"(reg), "r"((int)guard)
    );
}

__device__ __forceinline__
void lds128(float &reg0, float &reg1,
            float &reg2, float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
        : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
        : "r"(addr)
    );
}

__device__ __forceinline__
void sts32(const float &reg, const uint32_t &addr) {
    asm volatile (
        "st.shared.f32 [%0], %1;\n"
        : : "r"(addr), "f"(reg)
    );
}

__device__ __forceinline__
void sts128(const float &reg0, const float &reg1,
            const float &reg2, const float &reg3,
            const uint32_t &addr) {
    asm volatile (
        "st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
        : : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3)
    );
}

struct StgFrag {
    float data[4][4];

    __device__ __forceinline__
    StgFrag(const float (&C_frag)[8][8], int tile_x, int tile_y) {
        #pragma unroll  // ys: unroll the loops
        for (int i = 0; i < 4; ++i) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                data[i][j] = C_frag[tile_y * 4 + i][tile_x * 4 + j];
            }
        }
    }
};

__device__ __noinline__
void C_tile_wb(StgFrag C_frag,
               float *C_stg_ptr,
               const float *C_lds_ptr,
               uint32_t C_sts_addr,
               uint32_t m,
               uint32_t n,
               uint32_t m_idx,
               uint32_t n_idx) {
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        sts128(C_frag.data[i][0],
               C_frag.data[i][1],
               C_frag.data[i][2],
               C_frag.data[i][3],
               C_sts_addr + i * 8 * sizeof(float4));
    }

    __syncthreads();

    uint32_t m_guard = m < m_idx ? 0 : m - m_idx;

    #pragma unroll
    for (int i = 0; i < 16; ++i) {
        stg32(C_lds_ptr[i * 32],
              C_stg_ptr + i * n,
              i < m_guard && n_idx < n);
    }
}

/*
 * matrix A, B and C: row-major
 *
 * mma block:
 * thread block tile: m128n128k8
 * warp tile: m32n64k8
 * thread tile: m8n8k8
 * thread fragment:
 *     matrixA: 8x1 FP32
 *     matrixB: 1x8 FP32
 *
 * ----------------------------------------------------------------
 * thread block tile map:
 *
 *                                128
 *                    --|---------------------|
 *             B_tile  8|                     |
 *                    --|---------------------|
 *
 *  A_tile   | 8 |      |    64    |
 *         --|---|    --|----------|----------|
 *           |   |    32|  warp_0  |  warp_1  |
 *           |   |    --|----------|----------|
 *           |   |      |  warp_2  |  warp_3  |
 *        128|   |      |----------|----------|
 *           |   |      |  warp_4  |  warp_5  |
 *           |   |      |----------|----------|
 *           |   |      |  warp_6  |  warp_7  |
 *         --|---|      |----------|----------|
 *
 * ----------------------------------------------------------------
 * warp tile map:
 *
 * 'z' thread map to avoid LDS.128 shared memory broadcast limitation.
 *
 *              |              32               ||
 *     B_frag --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 *             1|///|   |   |   |   |   |   |   ||///|   |   |   |   |   |   |   |
 *            --|---|---|---|---|---|---|---|---||---|---|---|---|---|---|---|---|
 * A_frag       | 4 |                           ||
 *    | 1 |                                     ||
 *  --|---|--   |---|---|---|---|---|---|---|---||---|---------------------------|
 *    |///|4    |t0 |t2 |t4 |t6 |t8 |t10|t12|t14||t0 |                           |
 *    |---|--   |---|---|---|---|---|---|---|---||---|                           |
 *    |   |     |t1 |t3 |t5 |t7 |t9 |t11|t13|t15||                               |
 *  16|---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t16|t18|t20|t22|t24|t26|t28|t30||                               |
 *    |---|     |---|---|---|---|---|---|---|---||                               |
 *    |   |     |t17|t19|t21|t23|t25|t27|t29|t31||                               |
 *  ==|===|=====|===|===|===|===|===|===|===|===||===|============================
 *    |///|     |t0 |                           ||t0 |                           |
 *    |---|     |---|                           ||---|                           |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |                               ||                               |
 *    |   |     |                               ||                               |
 *    |---|     |-------------------------------||-------------------------------|
 *
 */
__global__ __launch_bounds__(256, 2)
void sgemm_128x128x8_kernel(const float *A,
                            const float *B,
                            float *C,
                            uint32_t m,
                            uint32_t n,
                            uint32_t k,
                            uint32_t A_ldg_step,    // k * sizeof(float)
                            uint32_t B_ldg_step) {  // n * sizeof(float) * 8
    /*
     * matrix A & B thread block tile shared memory (double buffer)
     * matrix A: 132 * 8 * 4Byte/item * double buffer = 4.125KB * 2
     * matrix B: 128 * 8 * 4Byte/item * double buffer = 8KB
     *
     * for double buffer faster switch, A_smem requires 8KB * 2 shared memory
     * and 16KB aligned, B_smem should be 8KB aligned, then the double buffer
     * can be switched by only 1 xor instruction:
     *     (uint32_t &)A_smem ^= 0x2000;
     *     (uint32_t &)B_smem ^= 0x1000;
     */
    __shared__ __align__(16 * 1024) char smem[24 * 1024];    //ys: Align smem to 16K
    float *A_smem = reinterpret_cast<float *>(smem);         //ys: re-interpret content of smem in "float" pattern 
    float *B_smem = reinterpret_cast<float *>(smem + 16 * 1024); //ys: Shift 16KB

    // A, B and C register fragment
    float A_frag[2][8];
    float B_frag[2][8];
    float C_frag[8][8];
    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        #pragma unroll
        for (int j = 0; j < 8; ++j) {
            C_frag[i][j] = 0;
        }
    }
    // ys: // Clear the thread_tile: Task for a single thread

    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t warp_id = threadIdx.x / 32;

    // 4x8 threads each warp for FFMA
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);
    
    /* ys:
     * Threads in a warp would be like:
     *
     * 0  1  16 17
     * 2  3  18 19
     * 4  5  20 21
     * 6  7  22 23
     * 8  9  24 25
     * 10 11 26 27
     * 12 13 28 29
     * 14 15 30 31
     * 
     * But why?
     */


    // A_tile & B_tile ldg pointer
    // haotian: equivalent to <<< (m/128, n/128), (32, 8)>>>, each warp load 4 from DRAM to shared (tblock: 128x8).
    const char *A_ldg_ptr = (const char *)(
        A + (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8);
    
    // ys:
    // 1 block —— 128 A rows.
    // We have 8 cols in each K_tile, so (threadIdx.x % 8) is the col_idx of current thread
    // 8 threads will take 4 rows, so each thread is dealing with 4 elements in A.
    
    // haotian: equivalent to <<< (m/128, n/128), (8, 32)>>>, each warp load 4 from DRAM to shared.
    const char *B_ldg_ptr = (const char *)(
        B + (threadIdx.x / 32) * n + blockIdx.x * 128 + threadIdx.x % 32);

    // ys:
    // 1 block —— 128 B cols
    // 32 threads —— 1 B row 
    // **  (K_tiles = 8, and we have 256 threads, so 32 threads is assigned to each row of B_tile)
    // in each B row, there are 128 elements and 32 threads, so each thread is dealing with 4 elements as well.


    // ys: NOT UNDERSTAND!! Why 132 for A?

    // A_tile & B_tile sts/lds pointer   (sts:store to smem  lds:load from smem)
    // using uint32_t pointer for faster double buffer switch
    // haotian: looks like 8 x 132 shared memory (what is 8x4 for?)
    uint32_t A_sts_addr = smem_u32addr(
        A_smem + (threadIdx.x % 8) * a_const + (threadIdx.x / 8) * 4);
    // haotian: looks like 128 x 8 shared memory
    // ys: each thread get 4 Bytes(one uint32) of Smem.
    // ys: data in Mat A will be broadcast to 32 threads in a warp.

    uint32_t B_sts_addr = smem_u32addr(
        B_smem + (threadIdx.x / 32) * 128 + (threadIdx.x % 32));
    // haotian: mma_tid_x in [0, 7] (for B shared mem), 
    //          mma_tid_y in [0, 3] (for A shared mem), 
    //          each thread load 4 (strided) x 4 (contig), 
    //          there might be broadcast.
    
    // ys: 32 threads get 128 Bytes, each thread get 4 Byte for B
    // ys: But why each thread's addr only +1?
    // ys: what is strided and contig?
    
    // haotian: warp_id is 4x2. per warp 32x1 or 1x64
    uint32_t A_lds_addr = smem_u32addr(
        A_smem + (warp_id / 2) * 32 + mma_tid_y * 4);
    uint32_t B_lds_addr = smem_u32addr(
        B_smem + (warp_id % 2) * 64 + mma_tid_x * 4);


    // ldg_guard to avoid LDG out of bound
    
    // ys: EACH THREAD HAS ITS OWN LDG_GUARD!!!!
    // ys: In the last 8 threads, they may deal with less than 4 rows, we need to record the num of rows.
    uint32_t A_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int m_idx = blockIdx.y * 128 + threadIdx.x / 8 * 4 + i;
        if (m_idx < m) {
            A_ldg_guard |= (1u << i);
        }
    }

    // ys: In the last block, there might be less than 128 cols of B, 
    //     and (threadIdx.x % 32 + i * 32) is the col that thread will deal with.
    uint32_t B_ldg_guard = 0;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        int n_idx = blockIdx.x * 128 + threadIdx.x % 32 + i * 32;
        if (n_idx < n) {
            B_ldg_guard |= (1u << i);
        }
    }

    float A_ldg_reg[4];
    float B_ldg_reg[4];

    // 1'st A&B tile loaded before the k_tile loop
    uint32_t k_tiles = (k + 7) / 8 - 1; // ys: k is given (A:M*K), not a iterator.

    // load 1'st tile to shared memory. 
    // haotian: A load 128x1, B load 1x128, each thread load 4.
    // A stored as 1x128, B stored as 128x1
    // ys: The 1st tile is not complete (maybe less than 8 cols for A)
    {
        uint32_t first_k_tile = k - k_tiles * 8; // ys: first_k_tile ranges from 1 to 8

        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            bool guard = (A_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x % 8 < first_k_tile;
            ldg32_nc_0(A_ldg_reg[i],
                       A_ldg_ptr + i * A_ldg_step,  // A_ldg_step =  k * sizeof(float);  jump to the next row of A
                       guard);
        }

        sts128(A_ldg_reg[0], A_ldg_reg[1], A_ldg_reg[2], A_ldg_reg[3],
               A_sts_addr);
        // ys: store continuous 128 bits to smem

        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            bool guard = (B_ldg_guard & (1u << i)) != 0 &&
                         threadIdx.x / 32 < first_k_tile;
            ldg32_nc_0(B_ldg_reg[i],    
                       B_ldg_ptr + i * 32 * sizeof(float),
                       guard);
        }

        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            sts32(B_ldg_reg[i], B_sts_addr + i * 32 * sizeof(float));
        }
        // ys: store 4*32 bits to smem

        __syncthreads();

        // switch double buffer
        A_sts_addr ^= 0x2000;   // ys: add/minus 8KB
        B_sts_addr ^= 0x1000;   // ys: add/minus 4KB
        // A_sts_addr is shifted from A_smem (No more than 4KB)
        // B_sts_addr is shifted from B_smem (No more than 4KB)
        // there is 16KB from A_smem to B_smem
        // Thus, it will work.
        // (A needs 8KB because the author is computing with 132*8*4 = 4.125KB)
 

        // ldg pointer for next tile
        A_ldg_ptr += first_k_tile * sizeof(float);
        B_ldg_ptr += n * first_k_tile * sizeof(float);
        // shift the ptr so that it will be the multiple of 8
    }

    // load 1'st fragment
    lds128(A_frag[0][0], A_frag[0][1], A_frag[0][2], A_frag[0][3],
           A_lds_addr);
    lds128(A_frag[0][4], A_frag[0][5], A_frag[0][6], A_frag[0][7],
           A_lds_addr + 16 * sizeof(float));
    lds128(B_frag[0][0], B_frag[0][1], B_frag[0][2], B_frag[0][3],
           B_lds_addr);
    lds128(B_frag[0][4], B_frag[0][5], B_frag[0][6], B_frag[0][7],
           B_lds_addr + 32 * sizeof(float));

    // k_tiles loop
    for (; k_tiles > 0; --k_tiles) {
        #pragma unroll
        for (int k_frag = 0; k_frag < 8; ++k_frag) {
            // store next A&B tile to shared memory
            // Haotian: note: there is one tile not used! so we still need an epilog.
            if (k_frag == 7) {
                sts128(A_ldg_reg[0], A_ldg_reg[1], A_ldg_reg[2], A_ldg_reg[3],
                       A_sts_addr);
                #pragma unroll
                for (int i = 0; i < 4; ++i) {
                    sts32(B_ldg_reg[i], B_sts_addr + i * 32 * sizeof(float));
                }

                __syncthreads();

                // switch double buffer
                A_lds_addr ^= 0x2000;
                B_lds_addr ^= 0x1000;
                A_sts_addr ^= 0x2000;
                B_sts_addr ^= 0x1000;

                // ldg pointer for next tile
                A_ldg_ptr += 8 * sizeof(float);
                B_ldg_ptr += B_ldg_step; // B_ldg_step = n * sizeof(float) * 8; jump 8 rows 
            }

            // load next A&B fragment from shared memory to register. Haotian: naturally double buffer (k_flag switches between even and odd)
            // +16: totally load 32 on A_frag dimension (load 64 on B_frag dimension). Half is 16 or 32.
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % 8 * a_const * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % 8 * a_const + 16) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float));

            // load next A&B tile. Haotian: 4 = 128 (tblock tile size) / 32 ??
            // ys: I agree. Since we have 256 threads, and K_tile = 8,
            //              4 = 128 / (256/8)
            if (k_frag == 0) {
                #pragma unroll
                for (int i = 0; i < 4; ++i) {
                    ldg32_nc(A_ldg_reg[i],
                             A_ldg_ptr + i * A_ldg_step,
                             (A_ldg_guard & (1u << i)) != 0);
                }

                #pragma unroll
                for (int i = 0; i < 4; ++i) {
                    ldg32_nc(B_ldg_reg[i],
                             B_ldg_ptr + i * 32 * sizeof(float),
                             (B_ldg_guard & (1u << i)) != 0);
                }
            }

            // FFMA loop (Outer Product)
            #pragma unroll
            for (int i = 0; i < 8; ++i) {
                #pragma unroll
                for (int j = 0; j < 8; ++j) {
                    C_frag[i][j] += A_frag[k_frag % 2][i] *
                                    B_frag[k_frag % 2][j];
                }
            }
        }
    }

    // FFMA for the last tile
    #pragma unroll
    for (int k_frag = 0; k_frag < 8; ++k_frag) {
        if (k_frag < 7) {
            // load next A&B fragment from shared memory to register
            lds128(A_frag[(k_frag + 1) % 2][0],
                   A_frag[(k_frag + 1) % 2][1],
                   A_frag[(k_frag + 1) % 2][2],
                   A_frag[(k_frag + 1) % 2][3],
                   A_lds_addr + (k_frag + 1) % 8 * a_const * sizeof(float));
            lds128(A_frag[(k_frag + 1) % 2][4],
                   A_frag[(k_frag + 1) % 2][5],
                   A_frag[(k_frag + 1) % 2][6],
                   A_frag[(k_frag + 1) % 2][7],
                   A_lds_addr + ((k_frag + 1) % 8 * a_const + 16) * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][0],
                   B_frag[(k_frag + 1) % 2][1],
                   B_frag[(k_frag + 1) % 2][2],
                   B_frag[(k_frag + 1) % 2][3],
                   B_lds_addr + (k_frag + 1) % 8 * 128 * sizeof(float));
            lds128(B_frag[(k_frag + 1) % 2][4],
                   B_frag[(k_frag + 1) % 2][5],
                   B_frag[(k_frag + 1) % 2][6],
                   B_frag[(k_frag + 1) % 2][7],
                   B_lds_addr + ((k_frag + 1) % 8 * 128 + 32) * sizeof(float));
        }

        // FFMA loop
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            #pragma unroll
            for (int j = 0; j < 8; ++j) {
                C_frag[i][j] += A_frag[k_frag % 2][i] *
                                B_frag[k_frag % 2][j];
            }
        }
    }


    // C_tile write back, reuse A&B tile shared memory buffer
    // haotian: mma_tid_x <= 8, mma_tid_y <= 4, per thread loads 4 data
    // ys: float4 combines 4 floats. 
    uint32_t C_sts_addr = smem_u32addr((float4 *)(smem + warp_id * 2048) +
                                       mma_tid_y * 4 * 8 + mma_tid_x);
    const float *C_lds_ptr = (float *)(smem + warp_id * 2048) + lane_id;

    uint32_t m_idx = blockIdx.y * 128 + warp_id / 2 * 32;
    uint32_t n_idx = blockIdx.x * 128 + warp_id % 2 * 64 + lane_id;

    float *C_stg_ptr = C + m_idx * n + n_idx;
    
    if (m_idx >= m) {
        return;
    } else if (m_idx + 32 <= m) {
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;

        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts128(C_frag[i * 4 + p][j * 4],
                           C_frag[i * 4 + p][j * 4 + 1],
                           C_frag[i * 4 + p][j * 4 + 2],
                           C_frag[i * 4 + p][j * 4 + 3],
                           C_sts_addr + p * 8 * sizeof(float4));
                }

                __syncthreads();
                #pragma unroll
                for (int p = 0; p < 16; ++p) {
                    stg32(C_lds_ptr[p * 32],
                          C_stg_ptr + (i * 16 + p) * n + j * 32,
                          j * 32 < n_guard);
                }
            }
        }
    } else {
        /*
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                StgFrag stg_frag(C_frag, j, i);

                C_tile_wb(stg_frag,
                          C_stg_ptr + i * 16 * n + j * 32,
                          C_lds_ptr,
                          C_sts_addr,
                          m,
                          n,
                          m_idx + i * 16,
                          n_idx + j * 32);
            }
        }
        */
        uint32_t n_guard = n < n_idx ? 0 : n - n_idx;
        uint32_t m_guard = m < m_idx ? 0 : m - m_idx;

        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                __syncthreads();

                #pragma unroll
                for (int p = 0; p < 4; ++p) {
                    sts128(C_frag[i * 4 + p][j * 4],
                           C_frag[i * 4 + p][j * 4 + 1],
                           C_frag[i * 4 + p][j * 4 + 2],
                           C_frag[i * 4 + p][j * 4 + 3],
                           C_sts_addr + p * 8 * sizeof(float4));
                }

                __syncthreads();
                #pragma unroll
                for (int p = 0; p < 16; ++p) {
                    stg32(C_lds_ptr[p * 32],
                          C_stg_ptr + (i * 16 + p) * n + j * 32,
                          i * 16 < m_guard && j * 32 < n_guard);
                }
            }
        }
    }
}

int main() {
    
    int m = 5120;
    int n = 4096;
    int k = 4096;
    
    /*
    int m = 5000;
    int n = 128;
    int k = 128;
    */
    int n_iter = 10;

    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, m * k * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, k * n * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, m * n * sizeof(float), hipHostMallocDefault);
    random_init(h_A, m * k);
    random_init(h_B, k * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyDefault);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 grid((n + 127) / 128, (m + 127) / 128);

    // warmup
    sgemm_128x128x8_kernel<<<grid, 256>>>(
        d_A, d_B, d_C, m, n, k, k * sizeof(float), n * sizeof(float) * 8);

    hipEventRecord(start);
    for (int i = 0; i < n_iter; ++i) {
        sgemm_128x128x8_kernel<<<grid, 256>>>(
            d_A, d_B, d_C, m, n, k, k * sizeof(float), n * sizeof(float) * 8);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    long workload = n_iter * long(m) * n * k * 2;
    double gflops = (double(workload) / 1e9) / (double(ms) / 1e3);
    printf("Performance: %fGFLOPS\n", gflops);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDefault);

    bool chk = check(h_A, h_B, h_C, m, n, k);
    printf("Matrix_C check: %s\n", chk ? "OK" : "Failed");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
}


